#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc) {
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    *acc = new int[rBins * degreeBins];
    memset(*acc, 0, sizeof(int) * rBins * degreeBins);
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    for (int i = 0; i < w; i++)
        for (int j = 0; j < h; j++) {
            int idx = j * w + i;
            if (pic[idx] > 0) {
                int xCoord = i - xCent;
                int yCoord = yCent - j;
                float theta = 0;
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    (*acc)[rIdx * degreeBins + tIdx]++;
                    theta += radInc;
                }
            }
        }
}

__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin) {
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h) return;

    int xCent = w / 2;
    int yCent = h / 2;
    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}

int main(int argc, char **argv) {
    PGMImage inImg(argv[1]);
    int *cpuht;
    int w = inImg.x_dim;
    int h = inImg.y_dim;

    float* d_Cos;
    float* d_Sin;
    hipMalloc((void **) &d_Cos, sizeof(float) * degreeBins);
    hipMalloc((void **) &d_Sin, sizeof(float) * degreeBins);

    CPU_HoughTran(inImg.pixels, w, h, &cpuht);

    float *pcCos = (float *) malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *) malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (int i = 0; i < degreeBins; i++) {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;
    hipMemcpy(d_Cos, pcCos, sizeof(float) * degreeBins, hipMemcpyHostToDevice);
    hipMemcpy(d_Sin, pcSin, sizeof(float) * degreeBins, hipMemcpyHostToDevice);

    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;
    h_in = inImg.pixels;
    h_hough = (int *) malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **) &d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **) &d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    int blockNum = ceil(w * h / 256.0);

    // Crear los eventos paso 2
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);
    // Liberar los eventos paso 2 cambiar para que este luego de la barrera de sincronizacion
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tiempo de ejecución del kernel: %f sec\n", milliseconds);

    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    for (int i = 0; i < degreeBins * rBins; i++) {
        if (cpuht[i] != h_hough[i])
            printf("Calculation mismatch at: %i %i %i\n", i, cpuht[i], h_hough[i]);
    }
    printf("Done!\n");
    // Liberar los eventos paso 2
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Liberar la memoria asignada
    hipFree(d_Cos);
    hipFree(d_Sin);
    hipFree(d_in);
    hipFree(d_hough);

    delete[] cpuht;
    free(pcCos);
    free(pcSin);
    free(h_hough);

    return 0;
}
